#include "hip/hip_runtime.h"
#include "CUBLAS_single.hpp"


__host__ void conjugate_gradient_blas(
    double* A, 
    double* x, 
    double* b, 
    int size,
    int max_iters, 
    double rel_error
)
{
    hipblasHandle_t cublasH = NULL;

    hipblasCreate(&cublasH);

    const int m = size;
    const int n = size;

    

    double alpha, beta, bb, rr, rr_new;
    int num_iters;


    // Vectors used on the GPU
    double* dev_A;
    double* dev_x;
    double* dev_b;
    double* dev_r;
    double* dev_p;
    double* dev_Ap;

    // Allocating vectors on GPU
    const unsigned int matrix_bytes = m * n * sizeof(double);
    const unsigned int vector_bytes = m * sizeof(double);

    hipMalloc(&dev_A, matrix_bytes);
    hipMalloc(&dev_x, vector_bytes);
    hipMalloc(&dev_b, vector_bytes);
    hipMalloc(&dev_r, vector_bytes);
    hipMalloc(&dev_p, vector_bytes);
    hipMalloc(&dev_Ap, vector_bytes);

    // Copying data from CPU to GPU
    hipMemcpy(dev_A, A, matrix_bytes, hipMemcpyHostToDevice);
    hipMemset(dev_x, 0, vector_bytes);
    hipMemcpy(dev_b, b, vector_bytes, hipMemcpyHostToDevice);
    hipMemcpy(dev_r, dev_b, vector_bytes, hipMemcpyDeviceToDevice);
    hipMemcpy(dev_p, dev_b, vector_bytes, hipMemcpyDeviceToDevice);


    // bb = dot(b, b, size);
    // Compute the dot product
    double one = 1.0;
    double zero = 0.0;
    double alpha2;
    double den;

    hipblasDdot(cublasH, m, dev_b, 1, dev_b, 1, &bb);
    
    rr = bb;
    for(num_iters = 1; num_iters <= max_iters; num_iters++)
    {

        hipblasDgemv(cublasH, HIPBLAS_OP_N, m, n, &one, dev_A, m, dev_p, 1, &zero, dev_Ap, 1);
        hipblasDdot(cublasH, m, dev_p, 1, dev_Ap, 1, &den);
        alpha = rr / den;

        // axpby(alpha, p, 1.0, x, size);
        hipblasDaxpy(cublasH, m, &alpha, dev_p, 1, dev_x, 1);
        

        // axpby(-alpha, Ap, 1.0, r, size);
        alpha2 = -alpha;
        hipblasDaxpy(cublasH, m, &alpha2, dev_Ap, 1, dev_r, 1);

        hipblasDdot(cublasH, m, dev_r, 1, dev_r, 1, &rr_new);

        beta = rr_new / rr;
        rr = rr_new;

        if(std::sqrt(rr / bb) < rel_error) { break; }
        // axpby(1.0, r, beta, p, size);
        hipblasDscal(cublasH, m, &beta, dev_p, 1);
        hipblasDaxpy(cublasH, m, &one, dev_r, 1, dev_p, 1);
    }

    if(num_iters <= max_iters)
    {
        //printf("Converged in %d iterations, relative error is %e\n", num_iters, std::sqrt(rr / bb));
        hipMemcpy(x, dev_x, vector_bytes, hipMemcpyDeviceToHost);
    }
    else
    {
        printf("Did not converge in %d iterations, relative error is %e\n", max_iters, std::sqrt(rr / bb));
    }

    hipFree(dev_A);
    hipFree(dev_x);
    hipFree(dev_b);
    hipFree(dev_r);
    hipFree(dev_p);
    hipFree(dev_Ap);
}